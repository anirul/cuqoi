#include "cuqoi.hpp"

#include <cassert>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdexcept>
#include <vector>

namespace cuqoi
{

CuqoiImage::CuqoiImage(const std::string& file_name)
{
    std::ifstream ifs(file_name, std::ios::binary);
    if (!ifs.is_open())
    {
        throw std::runtime_error("Could not open file");
    }

    // Determine the file's size
    ifs.seekg(0, std::ios::end);
    std::streamsize file_size = ifs.tellg();
    ifs.seekg(0, std::ios::beg);

    // Create a buffer to hold the file content; we use a vector of characters
    // for convenience
    std::vector<char> buffer(file_size);

    // Read the content of the file into the buffer
    if (!ifs.read(buffer.data(), file_size))
    {
        throw std::runtime_error("Could not read file.");
    }

    OpenContent(buffer.data(), file_size);
}

CuqoiImage::CuqoiImage(void* ptr, size_t size)
{
    throw std::runtime_error("Not implemented");
}

void CuqoiImage::OpenContent(void* ptr, size_t size)
{
    // Open the header_.
    if (header_size_ > size) 
    {
        throw std::runtime_error("Invalid header (image is too small).");
    }
    // Get the header from the data.
    std::memcpy(&header_, ptr, header_size_);
    if ((header_.magic[0] != 'q') || (header_.magic[1] != 'o') ||
        (header_.magic[2] != 'i') || (header_.magic[3] != 'f'))
    {
        throw std::runtime_error("Invalid header (invalid code).");
    }
    
    throw std::runtime_error("Not implemented");
}

cuqoi::CuqoiImage::~CuqoiImage()
{
    // TODO add a cuda free here?
}

void CuqoiImage::Decompress()
{
    throw std::runtime_error("Not implemented");
}

void CuqoiImage::Compress()
{
    throw std::runtime_error("Not implemented");
}

void* CuqoiImage::Data()
{
    throw std::runtime_error("Not implemented");
}

size_t CuqoiImage::Size() const
{
    throw std::runtime_error("Not implemented");
}

} // namespace cuqoi