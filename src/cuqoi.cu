#include "hip/hip_runtime.h"
#include "cuqoi.hpp"

#include <cassert>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdexcept>
#include <vector>

// This is the entry to the QOI byte decoding.
__global__ void DecryptQOIByte(
    int n, cuqoi::qoi_byte_storage_t* ptr, cuqoi::qoi_header_t header)
{
    int index = threadIdx.x;
    for (int i = 1; i < header.channels + 1; ++i)
    {
        if (header.channels == 3)
        {
            if (ptr[index - i].type !=
                cuqoi::qoi_type_t::QOI_OP_RGB)
            {
                ptr[index].valid = false;
            }
        }
        if (header.channels == 4)
        {
            if (ptr[index - i].type !=
                cuqoi::qoi_type_t::QOI_OP_RGBA)
            {
                ptr[index].valid = false;
            }
        }
    }
    if (!ptr[index].valid)
    {
        return;
    }
}

namespace cuqoi
{

CuqoiImage::CuqoiImage(const std::string& file_name)
{
    std::ifstream ifs(file_name, std::ios::binary);
    if (!ifs.is_open())
    {
        throw std::runtime_error("Could not open file");
    }

    // Determine the file's size
    ifs.seekg(0, std::ios::end);
    std::streamsize file_size = ifs.tellg();
    ifs.seekg(0, std::ios::beg);

    // Create a buffer to hold the file content; we use a vector of characters
    // for convenience
    file_buffer_.reserve(file_size);

    // Read the content of the file into the buffer
    if (!ifs.read(file_buffer_.data(), file_size))
    {
        throw std::runtime_error("Could not read file.");
    }
    // Check size (shouldn't happen).
    if (file_buffer_.size() != file_size)
    {
        throw std::runtime_error("Error in size.");
    }
    // Check the end of file.
    if (file_buffer_[file_size - 2] != 0x00 ||
        file_buffer_[file_size - 1] != 0x01)
    {
        throw std::runtime_error("No end, this should end by 0x00 && 0x01.");
    }

    OpenContent(file_buffer_.data(), file_size);
}

CuqoiImage::CuqoiImage(void* ptr, size_t size)
{
    throw std::runtime_error("Not implemented");
}

void CuqoiImage::OpenContent(void* ptr, size_t size)
{
    // Open the header_.
    if (QOI_HEADER_SIZE > size)
    {
        throw std::runtime_error("Invalid header (image is too small).");
    }
    // Get the header from the data.
    std::memcpy(&header_, ptr, QOI_HEADER_SIZE);
    if ((header_.magic[0] != 'q') || (header_.magic[1] != 'o') ||
        (header_.magic[2] != 'i') || (header_.magic[3] != 'f'))
    {
        throw std::runtime_error("Invalid header (invalid code).");
    }
    byte_size_ = size - (QOI_HEADER_SIZE + 2);
    if (byte_size_ <= 0)
    {
        throw std::runtime_error("Byte size shoulde be a valid number.");
    }
    // Allocate the CUDA memory.
    hipMalloc(&cuda_storage_, byte_size_);
    // Copy the data to the CUDA memory.
    void* buffer = &file_buffer_[QOI_HEADER_SIZE];
    hipMemcpy(&cuda_storage_, buffer, byte_size_, hipMemcpyHostToDevice);
}

cuqoi::CuqoiImage::~CuqoiImage()
{
    if (cuda_storage_)
    {
        hipFree(cuda_storage_);
    }
}

void CuqoiImage::Decrypt()
{
    DecryptQOIByte<<<1, byte_size_>>>(byte_size_, cuda_storage_, header_);
}

void CuqoiImage::Encrypt()
{
    throw std::runtime_error("Not implemented");
}

void* CuqoiImage::Data()
{
    throw std::runtime_error("Not implemented");
}

size_t CuqoiImage::Size() const
{
    throw std::runtime_error("Not implemented");
}

} // namespace cuqoi